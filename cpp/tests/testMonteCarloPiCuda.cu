#include "hip/hip_runtime.h"
#include "gaalet.h"
#include <iostream>
#include <cstdlib>

typedef gaalet::algebra<gaalet::signature<4,1> > cm;

__device__ float d_pi = 0.0;

__global__ void test()
{
   double r = (double)blockDim.x;
   
   unsigned int n_q = blockDim.x*blockDim.y*blockDim.z;
   __shared__ unsigned int n_s;
   if(threadIdx.x == 0 && threadIdx.y==0 && threadIdx.z==0) n_s = 0;

   cm::mv<0x01>::type e1(1.0);
   cm::mv<0x02>::type e2(1.0);
   cm::mv<0x04>::type e3(1.0);
   cm::mv<0x08>::type ep(1.0);
   cm::mv<0x10>::type em(1.0);

   cm::mv<0x00>::type one(1.0);

   cm::mv<0x08, 0x10>::type e0 = 0.5*(em-ep);
   cm::mv<0x08, 0x10>::type einf = em+ep;

   cm::mv<0x08, 0x10>::type S = e0 - 0.5*r*r*einf;


   cm::mv<0x01, 0x02, 0x04>::type x = ((double)threadIdx.x*e1 + (double)threadIdx.y*e2 + (double)threadIdx.z*e3)*r;
   cm::mv<0x01, 0x02, 0x04, 0x08, 0x10>::type P = x + 0.5*(x&x)*einf + e0;
   double d = eval(S&P);
   if(d>=0.0) {
      //atomicAdd(&n_s, 1);
      ++n_s;
   }
   __syncthreads();
   
   if(threadIdx.x == 0 && threadIdx.y==0 && threadIdx.z==0) d_pi = 6.0*(float)n_s/(float)n_q;
}


int main()
{
   std::cout << "Hello Gaalet Monte Carlo on Cuda!" << std::endl;


   dim3 threads( 1, 1, 1 );

   test <<< 1, threads >>>();

   float pi;
   if(hipMemcpyFromSymbol(&pi, HIP_SYMBOL("d_pi"), sizeof(pi), 0, hipMemcpyDeviceToHost) != hipSuccess) {
      std::cout << "hipMemcpyFromSymbol() not succeded!" << std::endl;
   }
   std::cout << "Pi: " << pi << std::endl;
   
   hipDeviceReset();
}
