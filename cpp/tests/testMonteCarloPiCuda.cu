#include "hip/hip_runtime.h"
#include "gaalet.h"
#include <iostream>
#include <cstdlib>

typedef gaalet::algebra<gaalet::signature<4,1> > cm;

__global__ void test()
{
   double r = (double)blockDim.x;
   
   unsigned int n_q = blockDim.x*blockDim.y*blockDim.z;
   __shared__ unsigned int n_s;
   if(threadIdx.x == 0 && threadIdx.y==0 && threadIdx.z==0) n_s = 0;

   cm::mv<0x01>::type e1(1.0);
   cm::mv<0x02>::type e2(1.0);
   cm::mv<0x04>::type e3(1.0);
   cm::mv<0x08>::type ep(1.0);
   cm::mv<0x10>::type em(1.0);

   cm::mv<0x00>::type one(1.0);

   cm::mv<0x08, 0x10>::type e0 = 0.5*(em-ep);
   cm::mv<0x08, 0x10>::type einf = em+ep;

   cm::mv<0x08, 0x10>::type S = eval(e0 - 0.5*r*r*einf);


   cm::mv<0x01, 0x02, 0x04>::type x = eval(((double)threadIdx.x*e1 + (double)threadIdx.y*e2 + (double)threadIdx.z*e3)*r);
   cm::mv<0x01, 0x02, 0x04, 0x08, 0x10>::type P = x + 0.5*(x&x)*einf + e0;
   double d = eval(S&P);
   if(d>=0.0) {
      atomicAdd(&n_s, 1);
   }
   __syncthreads();
}


int main()
{
   std::cout << "Hello Gaalet Monte Carlo on Cuda!" << std::endl;

   hipSetDevice( 0 );

   dim3 threads( 10, 10, 10 );

   test <<< 1, threads >>>();

   hipDeviceReset();

   //std::cout << "Pi: " << 6.0*(double)n_s/(double)n_q << std::endl;
}
