#include "hip/hip_runtime.h"
#include "gaalet.h"
#include <cmath>

int main()
{
   typedef gaalet::algebra<gaalet::signature<4,1> > cm;

   cm::mv<0x01>::type e1(1.0);
   cm::mv<0x02>::type e2(1.0);
   cm::mv<0x04>::type e3(1.0);
   cm::mv<0x08>::type ep(1.0);
   cm::mv<0x10>::type em(1.0);

   cm::mv<0x00>::type one(1.0);
   std::cout << "sin(one): " << sin(one) << std::endl;

   cm::mv<0x08, 0x10>::type e0 = 0.5*(em-ep);
   cm::mv<0x08, 0x10>::type einf = em+ep;

   cm::mv<0x18>::type E = ep*em;

   cm::mv<0x1f>::type I = e1*e2*e3*ep*em;
   //auto I_expr = e1*e2*e3*ep*em;
   //auto I_mv = eval(e1*e2*e3*ep*em);
   //std::cout << "I_expr: " << I_expr << ", I_mv[0]: " << I_mv[0] << std::endl;
   cm::mv<0x07>::type i = e1*e2*e3;

   std::cout << "e0*e0: " << e0*e0 << std::endl;
   std::cout << "einf*einf: " << einf*einf << std::endl;
   std::cout << "ep*ep: " << ep*ep << std::endl;
   std::cout << "em*em: " << em*em << std::endl;
   std::cout << "E: " << E << std::endl;
   std::cout << "ep*em: " << ep*em << std::endl;
   std::cout << "em*ep: " << em*ep << std::endl;
   std::cout << "e0*einf: " << e0*einf << std::endl;
   std::cout << "einf*e0: " << einf*e0 << std::endl;
   std::cout << "e0&einf: " << (e0&einf) << std::endl;
   std::cout << "einf&e0: " << (einf&e0) << std::endl;
   std::cout << "e0^einf: " << (e0^einf) << std::endl;
   std::cout << "einf^e0: " << (einf^e0) << std::endl;
}
