#include "hip/hip_runtime.h"
#include "gaalet.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <cmath>

typedef gaalet::algebra<gaalet::signature<3,0> > em;
typedef em::mv<0x01,0x02,0x04>::type Vector;
typedef em::mv<0x00,0x03,0x05,0x06>::type Rotor;

struct rotation_functor
{
   rotation_functor(const Rotor& setR)
      : R(setR),
        invR(!setR)
   { }

   __host__ __device__
      Vector operator()(const Vector& x) const
      { 
         return grade<1>(R*x*invR);
      }

   Rotor R;
   Rotor invR;
};


int main()
{
   thrust::host_vector<Vector> h_x(100);
   h_x[0][0] = 1.0; h_x[0][1] = 0.0; h_x[0][2] = 0.0;
   h_x[1][0] = 0.0; h_x[1][1] = 1.0; h_x[1][2] = 0.0;
   h_x[2][0] = 0.0; h_x[2][1] = 0.0; h_x[2][2] = 1.0;

   thrust::device_vector<Vector> d_x = h_x;

   thrust::device_vector<Vector> d_y(100);

   Rotor R;
   R[0] = cos(-0.5*0.5*M_PI); R[1] = sin(-0.5*0.5*M_PI);

   thrust::transform(d_x.begin(), d_x.end(), d_y.begin(), rotation_functor(R));

   thrust::host_vector<Vector> h_y = d_y;

   std::cout << "1: x: " << h_x[0] << ", y: " << h_y[0] << std::endl;
   std::cout << "2: x: " << h_x[1] << ", y: " << h_y[1] << std::endl;
   std::cout << "3: x: " << h_x[2] << ", y: " << h_y[2] << std::endl;
}
